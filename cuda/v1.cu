#include "hip/hip_runtime.h"
#include "common.hpp"

#include <fstream>
#include <cstdint>
#include <exception>
#include <vector>
#include <hiprand/hiprand_kernel.h>

using namespace std;

bool infer_mode = false;
string train_images_path;
string train_labels_path;
string val_images_path;
string val_labels_path;
string save_weights_path;
string load_weights_path;
int num_train_images;
int num_val_images;

int num_epochs = 10;
constexpr float learning_rate = 1e-6f;

constexpr int image_height = 28;
constexpr int image_width = 28;
constexpr int num_categories = 10;
constexpr int num_pixels_per_image = image_height * image_width;
constexpr int n_0 = num_pixels_per_image;
constexpr int n_1 = 128;
constexpr int n_2 = 128;
constexpr int n_3 = num_categories;
constexpr int num_weights = n_0 * n_1 + n_1 + n_1 * n_2 + n_2 + n_2 * n_3 + n_3;
constexpr int offset_w_1 = 0;
constexpr int offset_b_1 = offset_w_1 + n_0 * n_1;
constexpr int offset_w_2 = offset_b_1 + n_1;
constexpr int offset_b_2 = offset_w_2 + n_1 * n_2;
constexpr int offset_w_3 = offset_b_2 + n_2;
constexpr int offset_b_3 = offset_w_3 + n_2 * n_3;

float* d_w_1;
float* d_b_1;
float* d_z_1;
float* d_a_1;
float* d_w_2;
float* d_b_2;
float* d_z_2;
float* d_a_2;
float* d_w_3;
float* d_b_3;
float* d_z_3;
float* d_a_3;
float* d_grad_w_1;
float* d_grad_b_1;
float* d_grad_z_1;
float* d_grad_a_1_z_1;
float* d_grad_a_1;
float* d_grad_w_2;
float* d_grad_b_2;
float* d_grad_z_2;
float* d_grad_a_2_z_2;
float* d_grad_a_2;
float* d_grad_w_3;
float* d_grad_b_3;
float* d_grad_z_3;

float* d_z_1_infer;
float* d_a_1_infer;
float* d_z_2_infer;
float* d_a_2_infer;
float* d_z_3_infer;
float* d_a_3_infer;

float* h_loss_train;
float* d_loss_train;
int* h_count_correct_train;
int* d_count_correct_train;
float* h_loss_infer;
float* d_loss_infer;
int* h_count_correct_infer;
int* d_count_correct_infer;

void parseArguments(int argc, char* argv[]) {
    int i = 1;
    while (i < argc) {
        if (strcmp(argv[i], "--infer") == 0 || strcmp(argv[i], "-i") == 0) {
            infer_mode = true;
            i += 1;
        } else if (strcmp(argv[i], "--train-images") == 0) {
            train_images_path = argv[i + 1];
            i += 2;
        } else if (strcmp(argv[i], "--train-labels") == 0) {
            train_labels_path = argv[i + 1];
            i += 2;
        } else if (strcmp(argv[i], "--val-images") == 0) {
            val_images_path = argv[i + 1];
            i += 2;
        } else if (strcmp(argv[i], "--val-labels") == 0) {
            val_labels_path = argv[i + 1];
            i += 2;
        }else if (strcmp(argv[i], "--save-checkpoint") == 0) {
            save_weights_path = argv[i + 1];
            i += 2;
        } else if (strcmp(argv[i], "--load-checkpoint") == 0) {
            load_weights_path = argv[i + 1];
            i += 2;
        } else if (strcmp(argv[i], "--num-epochs") == 0) {
            num_epochs = atoi(argv[i + 1]);
            i += 2;
        } else {
            throw runtime_error("invalid arguments");
        }
    }
}

int32_t reverseInt32(uint8_t bytes[]) {
    return (bytes[0] << 24) | (bytes[1] << 16) | (bytes[2] << 8) | bytes[3];
}

uint8_t* readImagesIntoHostMemory(string& file_path, int& num_images) {
    ifstream file(file_path, ios::binary);
    if (!file.is_open()) {
        throw runtime_error("cannot open file " + file_path);
    }

    uint8_t buffer[16];
    file.read(reinterpret_cast<char*>(buffer), 16);
    int32_t magic_number = reverseInt32(buffer);
    num_images = reverseInt32(buffer + 4);
    int32_t read_image_height = reverseInt32(buffer + 8);
    int32_t read_image_width = reverseInt32(buffer + 12);
    if (magic_number != 0x803) {
        throw runtime_error("file contains invalid format - magic number " + magic_number);
    }
    if (read_image_height != image_height || read_image_width != image_width) {
        throw runtime_error("unexpected image size");
    }
    LOG("Found " << num_images << " images, size " << image_height << " x " << image_width << ".");

    int num_pixels_per_image = image_width * image_height;
    uint8_t* images = new uint8_t[num_images * num_pixels_per_image];

    for (int i = 0; i < num_images; ++i) {
        file.read(reinterpret_cast<char*>(&images[i * num_pixels_per_image]), num_pixels_per_image * sizeof(uint8_t));
    }

    file.close();
    return images;
}

uint8_t* readLabelsIntoHostMemory(string& file_path) {
    ifstream file(file_path, ios::binary);
    if (!file.is_open()) {
        throw runtime_error("cannot open file " + file_path);
    }

    uint8_t buffer[8];
    file.read(reinterpret_cast<char*>(buffer), 8);
    int32_t magic_number = reverseInt32(buffer);
    int32_t num_labels = reverseInt32(buffer + 4);
    if (magic_number != 0x801) {
        throw runtime_error("file contains invalid format - magic number " + magic_number);
    }
    LOG("Found " << num_labels << " labels.");

    uint8_t* onehot_labels = new uint8_t[num_labels * num_categories];
    uint8_t* labels = new uint8_t[num_labels];
    fill(onehot_labels, onehot_labels + num_labels * num_categories, 0);
    file.read(reinterpret_cast<char*>(labels), num_labels * sizeof(uint8_t));
    for (int i = 0; i < num_labels; ++i) {
        onehot_labels[i * num_categories + labels[i]] = 1;
    }

    file.close();
    delete[] labels;
    return onehot_labels;
}

void initData(string images_path, string labels_path, float*& d_images, float*& d_labels, int& num_images) {
    uint8_t* h_images = readImagesIntoHostMemory(images_path, num_images);
    uint8_t* h_labels = readLabelsIntoHostMemory(labels_path);

    float* h_images_pinned;
    float* h_labels_pinned;
    CHECK_CUDA(hipHostMalloc((void**)&h_images_pinned, num_images * num_pixels_per_image * sizeof(float)));
    CHECK_CUDA(hipHostMalloc((void**)&h_labels_pinned, num_images * num_categories * sizeof(float)));
    for (int i = 0; i < num_images * num_pixels_per_image; ++i) {
        h_images_pinned[i] = static_cast<float>(h_images[i]) / 255.0f;
    }
    for (int i = 0; i < num_images * num_categories; ++i) {
        h_labels_pinned[i] = static_cast<float>(h_labels[i]);
    }

    CHECK_CUDA(hipMalloc((void**)&d_images, num_images * num_pixels_per_image * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_labels, num_images * num_categories * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_images, h_images_pinned, num_images * num_pixels_per_image * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_labels, h_labels_pinned, num_images * num_categories * sizeof(float), hipMemcpyHostToDevice));
    LOG("Training data transfered to device memory.");

    CHECK_CUDA(hipHostFree(h_images_pinned));
    CHECK_CUDA(hipHostFree(h_labels_pinned));

    delete[] h_images;
    delete[] h_labels;
}

void saveWeights(string file_path) {
    ofstream file(file_path, ios::binary);
    if (!file.is_open()) {
        throw runtime_error("cannot open file " + file_path);
    }

    float* h_weights;
    CHECK_CUDA(hipHostMalloc((void**)&h_weights, num_weights * sizeof(float)));

    CHECK_CUDA(hipMemcpy((void*)(h_weights + offset_w_1), d_w_1, n_0 * n_1 * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy((void*)(h_weights + offset_b_1), d_b_1, n_1 * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy((void*)(h_weights + offset_w_2), d_w_2, n_1 * n_2 * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy((void*)(h_weights + offset_b_2), d_b_2, n_2 * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy((void*)(h_weights + offset_w_3), d_w_3, n_2 * n_3 * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy((void*)(h_weights + offset_b_3), d_b_3, n_3 * sizeof(float), hipMemcpyDeviceToHost));

    file.write(reinterpret_cast<char*>(h_weights), num_weights * sizeof(float));
    LOG("Weights saved to file.");

    CHECK_CUDA(hipHostFree(h_weights));
    file.close();
}

void loadWeights(string file_path) {
    ifstream file(file_path, ios::binary);
    if (!file.is_open()) {
        throw runtime_error("cannot open file " + file_path);
    }

    float* h_weights;
    CHECK_CUDA(hipHostMalloc((void**)&h_weights, num_weights * sizeof(float)));

    if (!file.read(reinterpret_cast<char*>(h_weights), num_weights * sizeof(float))) {
        CHECK_CUDA(hipFree(h_weights));
        file.close();
        throw runtime_error("cannot read file " + file_path);
    }
    LOG("Weights loaded from file into host memory.");

    CHECK_CUDA(hipMemcpy(d_w_1, (void*)(h_weights + offset_w_1), n_0 * n_1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b_1, (void*)(h_weights + offset_b_1), n_1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_w_2, (void*)(h_weights + offset_w_2), n_1 * n_2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b_2, (void*)(h_weights + offset_b_2), n_2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_w_3, (void*)(h_weights + offset_w_3), n_2 * n_3 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b_3, (void*)(h_weights + offset_b_3), n_3 * sizeof(float), hipMemcpyHostToDevice));
    LOG("Weights transfered from host to device memory.");

    CHECK_CUDA(hipHostFree(h_weights));
    file.close();
}

__global__ void g_transferAndConvertHTD(uint8_t* h_data, float* d_data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_data[idx] = static_cast<float>(h_data[idx]);
    }
}

__global__ void g_heWeightInitialization(float* d_weights, int m, int n, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_size = m * n;
    if (idx >= total_size) return;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    float random_normal = hiprand_normal(&state);
    float stddev = sqrtf(2.0f / (float)m);
    d_weights[idx] = random_normal * stddev;
}

__global__ void g_randomizeValues(float* a, int n, unsigned long long seed = 666) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        a[idx] = hiprand_uniform(&state) * 2.0f - 1.0f;
    }
}

__global__ void g_mulMats(float* mat_a, float* mat_b, float* mat_out, int m, int n, int k) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    float out_rc = 0;
    if (r < m && c < k) {
        for (int i = 0; i < n; ++i) {
            out_rc += mat_a[r * n + i] * mat_b[i * k + c];
        }
        mat_out[r * k + c] = out_rc;
    }
}

__global__ void g_addRowsMatVec(float* mat_a, float* vec_b, int m, int n) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (r < m && c < n) {
        mat_a[r * n + c] += vec_b[c];
    }
}

__global__ void g_activReLU(float* mat_in, float* mat_out, int m, int n) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (r < m && c < n) {
        mat_out[r * n + c] = max(mat_in[r * n + c], 0.0f);
    }
}

__global__ void g_activSoftmax(float* mat_in, float* mat_out, int m, int n) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (r >= m) return;

    float maxVal = -INFINITY;
    float sumExp = 0.0;

    for (int c = 0; c < n; c++) {
        maxVal = fmaxf(maxVal, mat_in[r * n + c]);
    }

    for (int c = 0; c < n; c++) {
        mat_out[r * n + c] = expf(mat_in[r * n + c] - maxVal);
        sumExp += mat_out[r * n + c];
    }

    for (int c = 0; c < n; c++) {
        mat_out[r * n + c] /= sumExp;
        mat_out[r * n + c] = max(mat_out[r * n + c], 0.001f);
    }
}

__global__ void g_subRowsMats(float* mat_a, float* mat_b, float* mat_out, int m, int n) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (r < m && c < n) {
        mat_out[r * n + c] = mat_a[r * n + c] - mat_b[r * n + c];
    } 
}

__global__ void g_mulMatsFirstTransposed(float* mat_a, float* mat_b, float* mat_out, int m, int n, int k) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    float out_rc = 0;
    if (r < m && c < k) {
        for (int i = 0; i < n; ++i) {
            out_rc += mat_a[i * m + r] * mat_b[i * k + c];
        }
        mat_out[r * k + c] = out_rc;
    }
}

__global__ void g_mulMatsSecondTransposed(float* mat_a, float* mat_b, float* mat_out, int m, int n, int k) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    float out_rc = 0;
    if (r < m && c < k) {
        for (int i = 0; i < n; ++i) {
            out_rc += mat_a[r * n + i] * mat_b[c * n + i];
        }
        mat_out[r * k + c] = out_rc;
    }
}

// __global__ void g_sumColsMat(float* mat, float* vec_out, int m, int n) {
//     int r = blockIdx.y * blockDim.y * 2 + threadIdx.y;
//     int c = blockIdx.x * blockDim.x + threadIdx.x;
//     if (r >= m || c >= n) return;

//     float org_val_1 = mat[r * n + c];
//     float org_val_2 = (r + blockDim.y < m) ? mat[(r + blockDim.y) * n + c] : 0;
//     for (int stride = blockDim.y; stride >= 1; stride /= 2) {
//         if (threadIdx.y < stride && r + stride < m) {
//             mat[r * n + c] += mat[(r + stride) * n + c];
//         }
//         __syncthreads();
//     }
//     if (threadIdx.y == 0) {
//         atomicAdd(&vec_out[c], mat[blockIdx.y * blockDim.y * 2 * n]);
//     }
//     __syncthreads();
//     mat[r * n + c] = org_val_1;
//     if (r + blockDim.y < m) mat[(r + blockDim.y) * n + c] = org_val_2;
// }

__global__ void g_sumColsMat(float* mat, float* vec_out, int m, int n) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (c >= n) return;

    float sum = 0.0f;
    for (int i = 0; i < m; ++i) {
        sum += mat[i * n + c];
    }
    vec_out[c] = sum;
}

__global__ void g_mulMatsElemWise(float* mat_a, float* mat_b, float* mat_out, int m, int n) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (r < m && c < n) {
        mat_out[r * n + c] = mat_a[r * n + c] * mat_b[r * n + c];
    }
}

__global__ void g_computeDerivReLU(float* mat_in, float* mat_out, int m, int n) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < m && c < n) {
        mat_out[r * n + c] = (mat_in[r * n + c] > 0) ? 1 : 0;
    }
}

__global__ void g_addLinear(float* dst, float* amount, float alpha, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        dst[idx] += alpha * amount[idx];
    }
}

__global__ void g_computeCrossEntropy(float* y_pred, float* y_true, float* result, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows) {
        float sumImage = 0.0;
        for (int j = 0; j < cols; j++) {
            int index = row * cols + j;
            sumImage += y_true[index] * log(y_pred[index]);
        }
        result[row] = -sumImage;
    }
}

__global__ void g_computeAccuracy(float* y_pred, float* y_true, int* correct, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows) {
        int predicted_label = 0;
        float max_val = y_pred[row * cols];
        for (int j = 1; j < cols; j++) {
            int index = row * cols + j;
            if (y_pred[index] > max_val) {
                max_val = y_pred[index];
                predicted_label = j;
            }
        }
        if (y_true[row * cols + predicted_label] == 1.0) {
            atomicAdd(correct, 1);
        }
    }
}

void print(float* d_data, int m, int n) {
    float* h_data;
    CHECK_CUDA(hipHostMalloc((void**)&h_data, m * n * sizeof(float)));
    CHECK_CUDA(hipMemcpy(h_data, d_data, m * n * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%f ", h_data[i * n + j]);
        }
        printf("\n");
    }
    CHECK_CUDA(hipHostFree(h_data));
}

void train() {
    float* d_train_images;
    float* d_train_labels;
    float* d_val_images;
    float* d_val_labels;

    initData(train_images_path, train_labels_path, d_train_images, d_train_labels, num_train_images);
    initData(val_images_path, val_labels_path, d_val_images, d_val_labels, num_val_images);
    LOG("Data initialized.");
    
    int n = num_train_images, n_infer = num_val_images;
    CHECK_CUDA(hipMalloc((void**)&d_w_1, n_0 * n_1 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_b_1, n_1 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_z_1, n * n_1 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_a_1, n * n_1 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_w_2, n_1 * n_2 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_b_2, n_2 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_z_2, n * n_2 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_a_2, n * n_2 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_w_3, n_2 * n_3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_b_3, n_3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_z_3, n * n_3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_a_3, n * n_3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_z_1_infer, n_infer * n_1 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_a_1_infer, n_infer * n_1 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_z_2_infer, n_infer * n_2 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_a_2_infer, n_infer * n_2 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_z_3_infer, n_infer * n_3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_a_3_infer, n_infer * n_3 * sizeof(float)));

    CHECK_CUDA(hipMalloc((void**)&d_grad_w_1, n_0 * n_1 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_grad_b_1, n_1 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_grad_z_1, n * n_1 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_grad_a_1_z_1, n * n_1 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_grad_a_1, n * n_1 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_grad_w_2, n_1 * n_2 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_grad_b_2, n_2 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_grad_z_2, n * n_2 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_grad_a_2_z_2, n * n_2 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_grad_a_2, n * n_2 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_grad_w_3, n_2 * n_3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_grad_b_3, n_3 * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_grad_z_3, n * n_3 * sizeof(float)));

    CHECK_CUDA(hipHostMalloc((void**)&h_loss_train, n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_loss_train, n * sizeof(float)));
    CHECK_CUDA(hipHostMalloc((void**)&h_count_correct_train, sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_count_correct_train, sizeof(int)));
    CHECK_CUDA(hipHostMalloc((void**)&h_loss_infer, n_infer * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_loss_infer, n_infer * sizeof(float)));
    CHECK_CUDA(hipHostMalloc((void**)&h_count_correct_infer, sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_count_correct_infer, sizeof(int)));

    unsigned long long random_seed = 666;
    {
        dim3 block_size(DEFAULT_BLOCKSIZE);
        dim3 grid_size((n_0 * n_1 + block_size.x - 1) / block_size.x);
        // g_randomizeValues<<<grid_size, block_size>>>(d_w_1, n_0 * n_1, random_seed);
        g_heWeightInitialization<<<grid_size, block_size>>>(d_w_1, n_0, n_1, random_seed);
    }
    {
        dim3 block_size(DEFAULT_BLOCKSIZE);
        dim3 grid_size((n_1 + block_size.x - 1) / block_size.x);
        g_randomizeValues<<<grid_size, block_size>>>(d_b_1, n_1, random_seed);
    }
    {
        dim3 block_size(DEFAULT_BLOCKSIZE);
        dim3 grid_size((n_1 * n_2 + block_size.x - 1) / block_size.x);
        // g_randomizeValues<<<grid_size, block_size>>>(d_w_2, n_1 * n_2, random_seed);
        g_heWeightInitialization<<<grid_size, block_size>>>(d_w_2, n_1, n_2, random_seed);

    }
    {
        dim3 block_size(DEFAULT_BLOCKSIZE);
        dim3 grid_size((n_2 + block_size.x - 1) / block_size.x);
        g_randomizeValues<<<grid_size, block_size>>>(d_b_2, n_2, random_seed);
    }
    {
        dim3 block_size(DEFAULT_BLOCKSIZE);
        dim3 grid_size((n_2 * n_3 + block_size.x - 1) / block_size.x);
        // g_randomizeValues<<<grid_size, block_size>>>(d_w_3, n_2 * n_3, random_seed);
        g_heWeightInitialization<<<grid_size, block_size>>>(d_w_3, n_2, n_3, random_seed);
    }
    {
        dim3 block_size(DEFAULT_BLOCKSIZE);
        dim3 grid_size((n_3 + block_size.x - 1) / block_size.x);
        g_randomizeValues<<<grid_size, block_size>>>(d_b_3, n_3, random_seed);
    }
    CHECK_CUDA(hipDeviceSynchronize());
    LOG("Weights initialized.");

    for (int epoch = 0; epoch < num_epochs; ++epoch) {
        // Forward
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_1 + block_size.x - 1) / block_size.x, (n + block_size.y - 1) / block_size.y);
            g_mulMats<<<grid_size, block_size>>>(d_train_images, d_w_1, d_z_1, n, n_0, n_1);
            g_addRowsMatVec<<<grid_size, block_size>>>(d_z_1, d_b_1, n, n_1);
            g_activReLU<<<grid_size, block_size>>>(d_z_1, d_a_1, n, n_1);
        }
        BREAK;
        LOG("Forwarded layer 1.");
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_2 + block_size.x - 1) / block_size.x, (n + block_size.y - 1) / block_size.y);
            g_mulMats<<<grid_size, block_size>>>(d_a_1, d_w_2, d_z_2, n, n_1, n_2);
            g_addRowsMatVec<<<grid_size, block_size>>>(d_z_2, d_b_2, n, n_2);
            g_activReLU<<<grid_size, block_size>>>(d_z_2, d_a_2, n, n_2);
            BREAK; print(d_a_2, 2, n_2);
        }
        BREAK;
        LOG("Forwarded layer 2.");
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_3 + block_size.x - 1) / block_size.x, (n + block_size.y - 1) / block_size.y);
            g_mulMats<<<grid_size, block_size>>>(d_a_2, d_w_3, d_z_3, n, n_2, n_3);
            g_addRowsMatVec<<<grid_size, block_size>>>(d_z_3, d_b_3, n, n_3);
        }
        BREAK;
        {
            dim3 block_size(1, DEFAULT_BLOCKSIZE);
            dim3 grid_size(1, (n + block_size.y - 1) / block_size.y);
            g_activSoftmax<<<grid_size, block_size>>>(d_z_3, d_a_3, n, n_3);
        }
        BREAK;
        print(d_z_3 + (n - 18) * n_3, 18, n_3);
        printf("---\n");
        print(d_a_3 + (n - 18) * n_3, 18, n_3);
        LOG("Forwarded layer 3.");
        // compute loss
        {
            float loss = 0.0f;
            dim3 block_size(DEFAULT_BLOCKSIZE);
            dim3 grid_size((n + block_size.x - 1) / block_size.x);
            g_computeCrossEntropy<<<grid_size, block_size>>>(d_a_3, d_train_labels, d_loss_train, n, n_3);
            CHECK_CUDA(hipMemcpy(h_loss_train, d_loss_train, n * sizeof(float), hipMemcpyDeviceToHost));
            print(d_loss_train + n - 18, 18, 1);
            for (int i = 0; i < n; ++i) {
                loss += h_loss_train[i];
            }
            LOG("Epoch " << epoch << " completed. Train Loss: " << loss);
        }

        //compute accuracy
        {
            float acc = 0.0f;
            CHECK_CUDA(hipMemset(d_count_correct_train, 0, sizeof(int)));
            dim3 block_size(DEFAULT_BLOCKSIZE);
            dim3 grid_size((n + block_size.x - 1) / block_size.x);
            g_computeAccuracy<<<grid_size, block_size>>>(d_z_3, d_train_labels, d_count_correct_train, n, n_3);
            CHECK_CUDA(hipMemcpy(h_count_correct_train, d_count_correct_train, sizeof(int), hipMemcpyDeviceToHost));
            CHECK_CUDA(hipDeviceSynchronize());
            acc = static_cast<float>(*h_count_correct_train) / n;
            LOG("Epoch " << epoch << " completed. Train Accuracy: " << acc);
        }

        // -------------------------------
        // Backward
        // L / z3
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_3 + block_size.x - 1) / block_size.x, (n + block_size.y - 1) / block_size.y);
            g_subRowsMats<<<grid_size, block_size>>>(d_a_3, d_train_labels, d_grad_z_3, n, n_3);
        }
        // CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("L/z3");
        // L / w3
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_3 + block_size.x - 1) / block_size.x, (n_2 + block_size.y - 1) / block_size.y);
            g_mulMatsFirstTransposed<<<grid_size, block_size>>>(d_a_2, d_grad_z_3, d_grad_w_3, n_2, n, n_3);
        }
        // CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        print(d_a_2, 1, n_2);
        print(d_grad_w_3, 1, n_3);
        LOG("L/w3");
        // L / b3
        // {
        //     dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
        //     dim3 grid_size((n_3 + block_size.x - 1) / block_size.x, (n + block_size.y * 2 - 1) / (block_size.y * 2));
        //     g_sumColsMat<<<grid_size, block_size>>>(d_grad_z_3, d_grad_b_3, n, n_3);
        // }
        {
            dim3 block_size(DEFAULT_BLOCKSIZE);
            dim3 grid_size((n_3 + block_size.x - 1) / block_size.x);
            g_sumColsMat<<<grid_size, block_size>>>(d_grad_z_3, d_grad_b_3, n, n_3);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("L/b3");
        // L / a2
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_2 + block_size.x - 1) / block_size.x, (n + block_size.y - 1) / block_size.y); 
            g_mulMatsSecondTransposed<<<grid_size, block_size>>>(d_grad_z_3, d_w_3, d_grad_a_2, n, n_3, n_2);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("L/a2");
        // a2 / z2
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_2 + block_size.x - 1) / block_size.x, (n + block_size.y - 1) / block_size.y);
            g_computeDerivReLU<<<grid_size, block_size>>>(d_a_2, d_grad_a_2_z_2, n, n_2);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("a2/z2");
        // L / z2
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_2 + block_size.x - 1) / block_size.x, (n + block_size.y - 1) / block_size.y);
            g_mulMatsElemWise<<<grid_size, block_size>>>(d_grad_a_2, d_grad_a_2_z_2, d_grad_z_2, n, n_2);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("L/z2");
        // L / w2
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_2 + block_size.x - 1) / block_size.x, (n_1 + block_size.y - 1) / block_size.y);
            g_mulMatsFirstTransposed<<<grid_size, block_size>>>(d_a_1, d_grad_z_2, d_grad_w_2, n_1, n, n_2);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("L/w2");
        // L / b2
        // {
        //     dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
        //     dim3 grid_size((n_2 + block_size.x - 1) / block_size.x, (n + block_size.y * 2 - 1) / (block_size.y * 2));
        //     g_sumColsMat<<<grid_size, block_size>>>(d_grad_z_2, d_grad_b_2, n, n_2);
        // }
        {
            dim3 block_size(DEFAULT_BLOCKSIZE);
            dim3 grid_size((n_2 + block_size.x - 1) / block_size.x);
            g_sumColsMat<<<grid_size, block_size>>>(d_grad_z_2, d_grad_b_2, n, n_2);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("L/b2");
        // L / a1
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_1 + block_size.x - 1) / block_size.x, (n + block_size.y - 1) / block_size.y); 
            g_mulMatsSecondTransposed<<<grid_size, block_size>>>(d_grad_z_2, d_w_2, d_grad_a_1, n, n_2, n_1);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("L/a1");
        // a1 / z1
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_1 + block_size.x - 1) / block_size.x, (n + block_size.y - 1) / block_size.y);
            g_computeDerivReLU<<<grid_size, block_size>>>(d_a_1, d_grad_a_1_z_1, n, n_1);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("a1/z1");
        // L / z1
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_1 + block_size.x - 1) / block_size.x, (n + block_size.y - 1) / block_size.y);
            g_mulMatsElemWise<<<grid_size, block_size>>>(d_grad_a_1, d_grad_a_1_z_1, d_grad_z_1, n, n_1);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("L/z1");
        // L / w1
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_1 + block_size.x - 1) / block_size.x, (n_0 + block_size.y - 1) / block_size.y);
            g_mulMatsFirstTransposed<<<grid_size, block_size>>>(d_train_images, d_grad_z_1, d_grad_w_1, n_0, n, n_1);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("L/w1");
        // L / b1
        // {
        //     dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
        //     dim3 grid_size((n_1 + block_size.x - 1) / block_size.x, (n + block_size.y * 2 - 1) / (block_size.y * 2));
        //     g_sumColsMat<<<grid_size, block_size>>>(d_grad_z_1, d_grad_b_1, n, n_1);
        // }
        {
            dim3 block_size(DEFAULT_BLOCKSIZE);
            dim3 grid_size((n_1 + block_size.x - 1) / block_size.x);
            g_sumColsMat<<<grid_size, block_size>>>(d_grad_z_1, d_grad_b_1, n, n_1);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("L/b1");

        // Update weight
        // w1
        {
            dim3 block_size(DEFAULT_BLOCKSIZE);
            dim3 grid_size((n_0 * n_1 + block_size.x - 1) / block_size.x);
            g_addLinear<<<grid_size, block_size>>>(d_w_1, d_grad_w_1, -learning_rate, n_0 * n_1);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("Update w1");
        // b1
        {
            dim3 block_size(DEFAULT_BLOCKSIZE);
            dim3 grid_size((n_1 + block_size.x - 1) / block_size.x);
            g_addLinear<<<grid_size, block_size>>>(d_b_1, d_grad_b_1, -learning_rate, n_1);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("Update b1");
        // w2
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_1 * n_2 + block_size.x - 1) / block_size.x);
            g_addLinear<<<grid_size, block_size>>>(d_w_2, d_grad_w_2, -learning_rate, n_1 * n_2);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("Update w2");
        // b2
        {
            dim3 block_size(DEFAULT_BLOCKSIZE);
            dim3 grid_size((n_2 + block_size.x - 1) / block_size.x);
            g_addLinear<<<grid_size, block_size>>>(d_b_2, d_grad_b_2, -learning_rate, n_2);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("Update b2");
        // w3
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_2 * n_3 + block_size.x - 1) / block_size.x);
            g_addLinear<<<grid_size, block_size>>>(d_w_3, d_grad_w_3, -learning_rate, n_2 * n_3);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("Update w3");
        // b3
        {
            dim3 block_size(DEFAULT_BLOCKSIZE);
            dim3 grid_size((n_3 + block_size.x - 1) / block_size.x);
            g_addLinear<<<grid_size, block_size>>>(d_b_3, d_grad_b_3, -learning_rate, n_3);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("Update b3");

        // Forward
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_1 + block_size.x - 1) / block_size.x, (n_infer + block_size.y - 1) / block_size.y);
            g_mulMats<<<grid_size, block_size>>>(d_val_images, d_w_1, d_z_1_infer, n_infer, n_0, n_1);
            g_addRowsMatVec<<<grid_size, block_size>>>(d_z_1_infer, d_b_1, n_infer, n_1);
            g_activReLU<<<grid_size, block_size>>>(d_z_1_infer, d_a_1_infer, n_infer, n_1);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("Forwarded layer 1.");
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_2 + block_size.x - 1) / block_size.x, (n_infer + block_size.y - 1) / block_size.y);
            g_mulMats<<<grid_size, block_size>>>(d_a_1_infer, d_w_2, d_z_2_infer, n_infer, n_1, n_2);
            g_addRowsMatVec<<<grid_size, block_size>>>(d_z_2_infer, d_b_2, n_infer, n_2);
            g_activReLU<<<grid_size, block_size>>>(d_z_2_infer, d_a_2_infer, n_infer, n_2);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("Forwarded layer 2.");
        {
            dim3 block_size(DEFAULT_TILEWIDTH, DEFAULT_TILEWIDTH);
            dim3 grid_size((n_3 + block_size.x - 1) / block_size.x, (n_infer + block_size.y - 1) / block_size.y);
            g_mulMats<<<grid_size, block_size>>>(d_a_2_infer, d_w_3, d_z_3_infer, n_infer, n_2, n_3);
            g_addRowsMatVec<<<grid_size, block_size>>>(d_z_3_infer, d_b_3, n_infer, n_3);
        }
        {
            dim3 block_size(1, DEFAULT_BLOCKSIZE);
            dim3 grid_size(1, (n_infer + block_size.y - 1) / block_size.y);
            g_activSoftmax<<<grid_size, block_size>>>(d_z_3_infer, d_a_3_infer, n_infer, n_3);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        BREAK;
        LOG("Forwarded layer 3.");

        // compute loss
        {
            float loss = 0.0f;
            dim3 block_size(DEFAULT_BLOCKSIZE);
            dim3 grid_size((n_infer + block_size.x - 1) / block_size.x);
            g_computeCrossEntropy<<<grid_size, block_size>>>(d_a_3_infer, d_val_labels, d_loss_infer, n_infer, n_3);
            CHECK_CUDA(hipMemcpy(h_loss_infer, d_loss_infer, n_infer * sizeof(float), hipMemcpyDeviceToHost));
            for (int i = 0; i < n; ++i) {
                loss += h_loss_infer[i];
            }
            LOG("Epoch " << epoch << " completed. Loss: " << loss);
        }

        //compute accuracy
        {
            float acc = 0.0f;
            CHECK_CUDA(hipMemset(d_count_correct_infer, 0, sizeof(int)));
            dim3 block_size(DEFAULT_BLOCKSIZE);
            dim3 grid_size((n_infer + block_size.x - 1) / block_size.x);
            g_computeAccuracy<<<grid_size, block_size>>>(d_a_3_infer, d_val_labels, d_count_correct_infer, n_infer, n_3);
            CHECK_CUDA(hipMemcpy(h_count_correct_infer, d_count_correct_infer, sizeof(int), hipMemcpyDeviceToHost));
            acc = static_cast<float>(*h_count_correct_infer) / n_infer;
            LOG("Epoch " << epoch << " completed. Accuracy: " << acc);
        }

        CHECK_CUDA(hipDeviceSynchronize());
        printf("Epoch %d completed.\n", epoch);
    }

    saveWeights(save_weights_path);

    CHECK_CUDA(hipFree(d_w_1));
    CHECK_CUDA(hipFree(d_b_1));
    CHECK_CUDA(hipFree(d_z_1));
    CHECK_CUDA(hipFree(d_a_1));
    CHECK_CUDA(hipFree(d_w_2));
    CHECK_CUDA(hipFree(d_b_2));
    CHECK_CUDA(hipFree(d_z_2));
    CHECK_CUDA(hipFree(d_a_2));
    CHECK_CUDA(hipFree(d_w_3));
    CHECK_CUDA(hipFree(d_b_3));
    CHECK_CUDA(hipFree(d_z_3));

    CHECK_CUDA(hipFree(d_grad_w_1));
    CHECK_CUDA(hipFree(d_grad_b_1));
    CHECK_CUDA(hipFree(d_grad_z_1));
    CHECK_CUDA(hipFree(d_grad_a_1_z_1));
    CHECK_CUDA(hipFree(d_grad_a_1));
    CHECK_CUDA(hipFree(d_grad_w_2));
    CHECK_CUDA(hipFree(d_grad_b_2));
    CHECK_CUDA(hipFree(d_grad_z_2));
    CHECK_CUDA(hipFree(d_grad_a_2_z_2));
    CHECK_CUDA(hipFree(d_grad_a_2));
    CHECK_CUDA(hipFree(d_grad_w_3));
    CHECK_CUDA(hipFree(d_grad_b_3));
    CHECK_CUDA(hipFree(d_grad_z_3));

    CHECK_CUDA(hipHostFree(h_loss_infer));
    CHECK_CUDA(hipFree(d_loss_infer));
    CHECK_CUDA(hipHostFree(h_count_correct_infer));
    CHECK_CUDA(hipFree(d_count_correct_infer));

    CHECK_CUDA(hipFree(d_train_images));
    CHECK_CUDA(hipFree(d_train_labels));
}

void infer() {

}

int main(int argc, char* argv[]) {
    parseArguments(argc, argv);
    LOG("parsed arguments");

    if (!infer_mode) {
        train();
    } else {
        infer();
    }

    return 0;
}
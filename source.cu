#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<string>
#include<math.h>
#define LEARNING_RATE 0.05
#define NUM_EPOCH 20
#define TRAIN_RATE 0.8
#define VAL_RATE 0.1
#define TEST_RATE 0.1

using namespace std;

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);                                                                 
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

int reverseInt (unsigned int i) 
{
    unsigned char c1, c2, c3, c4;

    c1 = i & 255;
    c2 = (i >> 8) & 255;
    c3 = (i >> 16) & 255;
    c4 = (i >> 24) & 255;

    return ((unsigned int)c1 << 24) + ((unsigned int)c2 << 16) + ((unsigned int)c3 << 8) + c4;
}

void read_mnist(string filename, double* &inputData, unsigned int& number_of_images, unsigned int& n_rows, unsigned int& n_cols)
{   
    ifstream file (filename);
    if (file.is_open())
    {
        unsigned int magic_number=0;
        file.read((char*)&magic_number,sizeof(magic_number)); 
        magic_number= reverseInt(magic_number);
        file.read((char*)&number_of_images,sizeof(number_of_images));
        number_of_images= reverseInt(number_of_images);
        file.read((char*)&n_rows,sizeof(n_rows));
        n_rows= reverseInt(n_rows);
        file.read((char*)&n_cols,sizeof(n_cols));
        n_cols= reverseInt(n_cols);
        cout << "Number of images: "<<number_of_images << endl;
        cout << "Number of rows: " << n_rows << endl;
        cout  << "Number of cols: " << n_cols << endl; 
        unsigned int required_mem_size = number_of_images * (n_rows * n_cols);
        inputData = (double*)malloc(required_mem_size * sizeof(double));
        for(int i=0;i<number_of_images;++i)
        {
            inputData[i*n_cols*n_rows] = 1;
            for(int r=0;r<n_rows;++r)
            {   
                for(int c=0;c<n_cols;++c)
                {
                    unsigned char temp=0;
                    file.read((char*)&temp,sizeof(temp));
                    // Doc tung pixel
                    inputData[i*n_rows*n_cols + r * n_cols + c] = double(temp) / 255;
                }

            }
        }
    }
}

void read_labels_one_hot(string filename, double* inputLabel) {
    ifstream file (filename);
    if (file.is_open()) {
        unsigned int magic_number = 0;
        unsigned int number_of_label = 0;
        file.read((char*)&magic_number, sizeof(magic_number));
        magic_number = reverseInt(magic_number);
        file.read((char*)&number_of_label, sizeof(number_of_label));
        number_of_label = reverseInt(number_of_label);
        cout << "Magic number: " << magic_number << endl;
        cout << "Number of label: " << number_of_label << endl;

        for (int i = 0; i < number_of_label; i++) {
            // One hot cua cac label
            unsigned char temp=0;
            file.read((char*)&temp,sizeof(temp));
            inputLabel[i*10 + int(temp)] = 1;
        }
    }
    file.close();
}

void read_labels(string filename, double* input_labels) {
    ifstream file (filename);
    if (file.is_open()) {
        unsigned int magic_number = 0;
        unsigned int number_of_label = 0;
        file.read((char*)&magic_number, sizeof(magic_number));
        magic_number = reverseInt(magic_number);
        file.read((char*)&number_of_label, sizeof(number_of_label));
        number_of_label = reverseInt(number_of_label);
        cout << "Magic number: " << magic_number << endl;
        cout << "Number of label: " << number_of_label << endl;

        for (int i = 0; i < number_of_label; i++) {
            // One hot cua cac label
            unsigned char temp=0;
            file.read((char*)&temp,sizeof(temp));
            input_labels[i] = temp;
        }
    }
    file.close();
}

void forwardNN(double* input, double* weight, double* bias, double* output, int inputRows, int inputCols, int outputCols, bool usedActivate = true) {
    for (int i = 0; i < inputRows; i++) {
        for (int j = 0; j < outputCols; j++) {
            double temp = 0;
            for (int k = 0; k < inputCols; k++) {
                temp += input[i * inputCols + k] * weight[k * outputCols + j];
            }
            temp += bias[j];
            if (usedActivate) {
                output[i * outputCols + j] = relu(temp);
            } else {
                output[i * outputCols + j] = temp;
            }
        }
    }
}

void softmax(double* input, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        double sum = 0.0;

        for (int j = 0; j < cols; j++) {
            double temp = exp(input[i * cols + j]);
            sum += temp;
            input[i*cols + j] = temp;
        }

        for (int j = 0; j < cols; j++) {
            input[i * cols + j] /= sum;
        }
    }
}

void initialize_weights(double* weights, int n_in, int n_out)  {
    double stddev = sqrt(2.0 / n_in); 
    for (int i = 0; i < n_in * n_out; i++) {
        double u1 = (double)rand() / RAND_MAX;
        double u2 = (double)rand() / RAND_MAX;
        double z = sqrt(-2.0 * log(u1)) * cos(2 * M_PI * u2); 
        weights[i] = z * stddev;
    }
}

void initialize_biases(double* bias, int n_in, int n_out) {
    double stddev = sqrt(2.0 / n_in); 
    for (int i = 0; i < n_out; i++) {
        double u1 = (double)rand() / RAND_MAX;
        double u2 = (double)rand() / RAND_MAX;
        double z = sqrt(-2.0 * log(u1)) * cos(2 * M_PI * u2); 
        bias[i] = z * stddev;
    }
}

void calculateLastDelta(double* y_pred, double* y, double* delta, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            delta[i*cols + j] = y_pred[i*cols + j] - y[i*cols + j];
        }
    }
}

void multiplyMatrix(double* matrix_a, double* matrix_b, double* result,int rows_a, int cols_a, int cols_b) {
    for (int i = 0; i< rows_a; i++) {
        for (int j = 0; j < cols_b; j++) {
            double temp = 0.0;
            for (int k = 0; k < cols_a; k++) {
                temp += matrix_a[i*cols_a + k] * matrix_b[k * cols_b + j];
            }
            result[i*cols_b + j] = temp;
        }
    }
}

void transposeMatrix(double* inputMatrix, double* outputMatrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            outputMatrix[j * rows + i] = inputMatrix[i * cols + j];
        }
    }
}

void relu_derivative(double* input, double* output, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if (input[i*cols + j] > 0) {
                output[i*cols + j] = 1;
            } else {
                output[i*cols + j] = 0;
            }
        }
    }
}

void multiplyMatrixElementWise(double* matrix_a, double* matrix_b, double* result, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            result[i * cols+ j] = matrix_a[i * cols + j] * matrix_b[i*cols + j];
        }
    }
}

void gradientForBias(double* delta, double* gradient,int rows, int cols) {
    for (int c = 0; c < cols; c++) {
        double temp = 0;
        for (int r = 0; r < rows; r++) {
            temp += delta[r*cols+c];
        }
        gradient[c] = temp / rows;
    }
}

void updateWeights(double* weights, double* gradient, double lr, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            weights[i * cols +j] -= lr * gradient[i*cols + j];
        }
    }
}

void updateBias(double* bias, double* gradient, double lr, int layerSize) {
    for (int i = 0; i < layerSize; i++) {
        bias[i] -= lr * gradient[i];
    }
}

double crossEntropy(double* y_pred, double* groundTruthOneHot, int rows, int cols) {
    double result = 0;

    for (int i = 0; i < rows; i++) {
        double sumImage = 0;
        for (int j = 0; j < cols; j++) {
            sumImage += groundTruthOneHot[i * cols + j]*log(y_pred[i * cols + j]);
        }
        result += -sumImage;
    }

    result /= rows;
    return result;
}

double accuracy(double* lastLayerResult, double* labels, int rows, int cols) {
    double count = 0;
    for (int r = 0; r < rows; r++) {
        double maxProp = lastLayerResult[r * cols];
        int label = 0;

        for (int c = 0; c < cols; c++) {
            if (maxProp < lastLayerResult[r * cols + c]) {
                maxProp = lastLayerResult[r * cols + c];
                label = c;
            }
        }

        if (labels[r] == double(label)) {
            count++;
        }
    }
    double result = count / rows;
    return result;
}

// shuffle data

void shuffle_data(double* data, double* one_hot_label, double* labels, int rows, int cols, int one_hot_cols) {
    for (int row = rows - 1; row > 0; row--) {
        int swap_row_index = rand() % (row + 1);

        for (int col = 0; col < cols; col++) {
            double temp = data[row * cols + col];
            data[row * cols + col] = data[swap_row_index * cols + col];
            data[swap_row_index * cols + col] = temp; 
        }

        // swap one hot label

        for (int col = 0; col < one_hot_cols; col++) {
            double temp = one_hot_label[row * one_hot_cols + col];
            one_hot_label[row * one_hot_cols + col] = one_hot_label[swap_row_index * one_hot_cols + col];
            one_hot_label[swap_row_index * one_hot_cols + col] = temp;
        }

        // swap label

        double temp = labels[row];
        labels[row] = labels[swap_row_index];
        labels[swap_row_index] = temp;
    }
}

// phan chia du lieu

void split(double* data, double* one_hot_labels, double* labels, int rows, int cols, int one_hot_cols, \
           double* train_data, double* train_one_hot, double* train_labels,\
            double* val_data, double* val_one_hot, double* val_labels,\
            double* test_data, double* test_one_hot, double* test_labels) {
    int train_limit = TRAIN_RATE * rows;
    int val_limit = (TRAIN_RATE + VAL_RATE) * rows;
    int row = 0,train_index = 0, val_index = 0, test_index = 0;

    while (row < train_limit) {
        // copy data anh
        for (int col = 0; col < cols; col++) {
            train_data[train_index * cols + col] = data[row * cols + col];
        }

        // copy data one hot label
        for (int col = 0; col < one_hot_cols; col++) {
            train_one_hot[train_index * one_hot_cols + col] = one_hot_labels[row * one_hot_cols + col];
        }

        // copy label

        train_labels[train_index] = labels[row];
        row++;
        train_index++;
    }
    
    while (row < val_limit) {
        // copy data anh
        for (int col = 0; col < cols; col++) {
            val_data[val_index * cols + col] = data[row * cols + col];
        }

        // copy data one hot label
        for (int col = 0; col < one_hot_cols; col++) {
            val_one_hot[val_index * one_hot_cols + col] = one_hot_labels[row * one_hot_cols + col];
        }

        // copy label

        val_labels[val_index] = labels[row];
        row++;
        val_index++;
    }

    while (row < rows) {
        // copy data anh
        for (int col = 0; col < cols; col++) {
            test_data[test_index * cols + col] = data[row * cols + col];
        }

        // copy data one hot label
        for (int col = 0; col < one_hot_cols; col++) {
            test_one_hot[test_index * one_hot_cols + col] = one_hot_labels[row * one_hot_cols + col];
        }

        // copy label

        test_labels[test_index] = labels[row];
        row++;
        test_index++;
    }
}

void backwardNN(double* transposedMatrix, double* delta, double* gradient, int inHiddenLayerSize, int numSample, int outHiddenLayerSize) {
    for (int row = 0; row < inHiddenLayerSize; row++) {
        for (int col = 0; col < outHiddenLayerSize; col++) {
            double temp = 0;

            for (int index  = 0; index < numSample; index++) {
                temp += transposedMatrix[row * numSample + index] * delta[index * outHiddenLayerSize + col];
            }

            gradient[row * outHiddenLayerSize + col] = temp / numSample;
        }
    }
}

void trainNN(double* train_data, double* train_one_hot_labels, double* train_labels, double* val_data, double* val_labels, double* val_one_hot_labels, double* firstHiddenLayerWeight, double *secondHiddenLayerWeight, double *lastHiddenLayerWeight, double* firstBiases, double *secondBiases, double *lastBiases, int num_epoch, int rows, int inputCols, int firstHiddenLayerSize, int secondHiddenLayerSize, int lastHiddenLayerSize) {
    double* firstLayerResult, *secondLayerResult, *lastLayerResult;
    double* transposeSecondResult, *transposeFirstResult, *transposeInputMatrix;
    double* lastDelta, *secondDelta, *firstDelta;
    double* lastGradient, *secondGradient, *firstGradient;
    double* transposeLastWeight, *transposeSecondWeight;
    double* reluDerivativeSecondMatrix, *reluDerivativeFirstMatrix;
    double* thirdBiasGradient, *secondBiasGradient, *firstBiasGradient;

    transposeInputMatrix = (double*)malloc(rows * inputCols * sizeof(double));
    firstLayerResult = (double*)malloc(rows * firstHiddenLayerSize * sizeof(double));
    secondLayerResult = (double*)malloc(rows * secondHiddenLayerSize * sizeof(double));
    lastLayerResult = (double*)malloc(rows * lastHiddenLayerSize * sizeof(double));

    transposeSecondResult = (double*)malloc(rows * secondHiddenLayerSize * sizeof(double));
    transposeFirstResult = (double*)malloc(rows * firstHiddenLayerSize * sizeof(double));

    lastDelta = (double*)malloc(rows * lastHiddenLayerSize * sizeof(double));
    secondDelta = (double*)malloc(rows * secondHiddenLayerSize * sizeof(double));
    firstDelta = (double*)malloc(rows * firstHiddenLayerSize * sizeof(double));

    lastGradient = (double*)malloc(secondHiddenLayerSize * lastHiddenLayerSize * sizeof(double));
    secondGradient = (double*)malloc(firstHiddenLayerSize * secondHiddenLayerSize * sizeof(double));
    firstGradient = (double*)malloc(inputCols * firstHiddenLayerSize * sizeof(double));

    transposeLastWeight = (double*)malloc(lastHiddenLayerSize * secondHiddenLayerSize * sizeof(double));
    transposeSecondWeight = (double*)malloc(secondHiddenLayerSize * firstHiddenLayerSize * sizeof(double));

    reluDerivativeSecondMatrix = (double*)malloc(rows * secondHiddenLayerSize * sizeof(double));
    reluDerivativeFirstMatrix = (double*)malloc(rows* firstHiddenLayerSize * sizeof(double));

    thirdBiasGradient = (double*)malloc(secondHiddenLayerSize  * lastHiddenLayerSize * sizeof(double));
    secondBiasGradient = (double*)malloc(firstHiddenLayerSize * secondHiddenLayerSize * sizeof(double));
    firstBiasGradient = (double*)malloc(inputCols * firstHiddenLayerSize * sizeof(double));

    transposeMatrix(train_data, transposeInputMatrix, rows, inputCols);

    for (int i = 0; i < num_epoch; i++) {
        // Forward qua 3 lop
        GpuTimer timer;
        timer.Start();

        forwardNN(train_data, firstHiddenLayerWeight, firstBiases, firstLayerResult, rows, inputCols, firstHiddenLayerSize);
        timer.Stop();
        float time = timer.Elapsed();
        printf("Thoi gian forward qua lop dau: %f \n", time);
        forwardNN(firstLayerResult, secondHiddenLayerWeight, secondBiases, secondLayerResult, rows, firstHiddenLayerSize, secondHiddenLayerSize);
        forwardNN(secondLayerResult, lastHiddenLayerWeight, lastBiases, lastLayerResult, rows, secondHiddenLayerSize, lastHiddenLayerSize, false);
        // Goi ham softmax cho ket qua cua layer cuoi
        timer.Start();
        softmax(lastLayerResult, rows, lastHiddenLayerSize);
        timer.Stop();
        time = timer.Elapsed();
        printf("Thoi gian softmax qua lop dau: %f \n", time);

        // backprop

        // Tinh transpose truoc
        timer.Start();
        transposeMatrix(secondLayerResult, transposeSecondResult, rows, secondHiddenLayerSize);
        timer.Stop();
        time = timer.Elapsed();
        printf("Thoi gian transpose thu hai: %f \n", time);
        timer.Start();
        transposeMatrix(firstLayerResult, transposeFirstResult, rows, firstHiddenLayerSize);
        timer.Stop();
        time = timer.Elapsed();
        printf("Thoi gian transpose thu nhat: %f \n", time);

        timer.Start();
        calculateLastDelta(lastLayerResult, train_one_hot_labels, lastDelta, rows, lastHiddenLayerSize);
        timer.Stop();
        time = timer.Elapsed();
        printf("Thoi gian tinh delta: %f \n", time);

        // Tinh cho gradient lop cuoi
        timer.Start();
        // multiplyMatrix(transposedSecondResult, lastDelta, lastGradient, secondHiddenLayerSize, numTrainSamples, lastHiddenLayerSize);
        // devideMatrixToScalar(lastGradient, numTrainSamples, secondHiddenLayerSize, lastHiddenLayerSize);
        backwardNN(transposeSecondResult, lastDelta, lastGradient, secondHiddenLayerSize, rows, lastHiddenLayerSize);
        gradientForBias(lastDelta, thirdBiasGradient, rows, lastHiddenLayerSize);
        timer.Stop();
        time = timer.Elapsed();
        printf("Tinh gradient cho lop cuoi: %f \n", time);

        timer.Start();
        relu_derivative(secondLayerResult, reluDerivativeSecondMatrix, rows, secondHiddenLayerSize);
        relu_derivative(firstLayerResult, reluDerivativeFirstMatrix, rows, firstHiddenLayerSize);
        timer.Stop();
        time = timer.Elapsed();
        printf("Tong thoi gian tinh dao ham relu: %f \n", time);


        // Cho hidden layer 2

        //tinh delta
        timer.Start();
        transposeMatrix(lastHiddenLayerWeight, transposeLastWeight, secondHiddenLayerSize, lastHiddenLayerSize);
        multiplyMatrix(lastDelta, transposeLastWeight, secondDelta, rows, lastHiddenLayerSize, secondHiddenLayerSize);
        multiplyMatrixElementWise(secondDelta, reluDerivativeSecondMatrix, secondDelta, rows, secondHiddenLayerSize);
        timer.Stop();
        time = timer.Elapsed();
        printf("Thoi gian tinh delta 2: %f \n", time);

        //tinh gradient 
        timer.Start();
        // multiplyMatrix(transposedFirstResult, secondDelta, secondGradient, firstHiddenLayerSize, numTrainSamples, secondHiddenLayerSize);
        // devideMatrixToScalar(secondGradient, numTrainSamples, firstHiddenLayerSize, secondHiddenLayerSize);
        backwardNN(transposeFirstResult, secondDelta, secondGradient, firstHiddenLayerSize, rows, secondHiddenLayerSize);
        gradientForBias(secondDelta, secondBiasGradient, rows, secondHiddenLayerSize);
        timer.Stop();
        time = timer.Elapsed();
        printf("Thoi gian tinh gradient cho lop thu hai: %f \n", time);
        

        // Cho hidden layer 1

        //tinh delta
        timer.Start();
        transposeMatrix(secondHiddenLayerWeight, transposeSecondWeight, firstHiddenLayerSize, secondHiddenLayerSize);
        multiplyMatrix(secondDelta, transposeSecondWeight, firstDelta, rows, secondHiddenLayerSize, firstHiddenLayerSize);
        multiplyMatrixElementWise(firstDelta, reluDerivativeFirstMatrix, firstDelta, rows, firstHiddenLayerSize);
        timer.Stop();
        time = timer.Elapsed();
        printf("Thoi gian tinh first delta: %f \n", time);

        // tinh gradient 
        // tinh chuyen vi cua ma tran dau vao
        timer.Start();
        // multiplyMatrix(transposedInputMatrix, firstDelta, firstGradient, inputLayerSize, numTrainSamples, firstHiddenLayerSize);
        // devideMatrixToScalar(firstGradient, numTrainSamples, inputLayerSize, firstHiddenLayerSize);
        backwardNN(transposeInputMatrix, firstDelta, firstGradient, inputCols, rows, firstHiddenLayerSize);
        gradientForBias(firstDelta, firstBiasGradient, rows, firstHiddenLayerSize);
        timer.Stop();
        time = timer.Elapsed();
        printf("Thoi gian tinh gradient cho lop dau: %f \n", time);
        

        timer.Start();
        // Cap nhat trong so
        // Layer 3
        updateWeights(lastHiddenLayerWeight, lastGradient, LEARNING_RATE, secondHiddenLayerSize, lastHiddenLayerSize);
        updateBias(lastBiases, thirdBiasGradient, LEARNING_RATE,lastHiddenLayerSize);

        // Layer 2
        updateWeights(secondHiddenLayerWeight, secondGradient, LEARNING_RATE, firstHiddenLayerSize, secondHiddenLayerSize);
        updateBias(secondBiases, secondBiasGradient, LEARNING_RATE,secondHiddenLayerSize);

        //layer 1
        updateWeights(firstHiddenLayerWeight, firstGradient, LEARNING_RATE, inputCols, firstHiddenLayerSize);
        updateBias(firstBiases, firstBiasGradient, LEARNING_RATE, firstHiddenLayerSize);
        timer.Stop();
        time = timer.Elapsed();
        cout << "Thoi gian cap nhat: " << time << " ms" << endl;

        forwardNN(train_data, firstHiddenLayerWeight, firstBiases, firstLayerResult, TRAIN_RATE * rows, inputCols, firstHiddenLayerSize);
        forwardNN(firstLayerResult, secondHiddenLayerWeight, secondBiases, secondLayerResult, TRAIN_RATE * rows, firstHiddenLayerSize, secondHiddenLayerSize);
        forwardNN(secondLayerResult, lastHiddenLayerWeight, lastBiases, lastLayerResult, TRAIN_RATE * rows, secondHiddenLayerSize, lastHiddenLayerSize, false);

        softmax(lastLayerResult, TRAIN_RATE * rows, lastHiddenLayerSize);

        cout << "Epoch: " << i ;
        cout <<", Train Accuracy: " << accuracy(lastLayerResult, train_labels, TRAIN_RATE * rows, lastHiddenLayerSize) << \
            ", Train Loss: " << crossEntropy(lastLayerResult, train_one_hot_labels, TRAIN_RATE * rows, lastHiddenLayerSize);

        forwardNN(val_data, firstHiddenLayerWeight, firstBiases, firstLayerResult, VAL_RATE * rows, inputCols, firstHiddenLayerSize);
        forwardNN(firstLayerResult, secondHiddenLayerWeight, secondBiases, secondLayerResult, VAL_RATE * rows, firstHiddenLayerSize, secondHiddenLayerSize);
        forwardNN(secondLayerResult, lastHiddenLayerWeight, lastBiases, lastLayerResult, VAL_RATE * rows, secondHiddenLayerSize, lastHiddenLayerSize, false);

        softmax(lastLayerResult, VAL_RATE * rows, lastHiddenLayerSize);

        cout <<", Val Accuracy: " << accuracy(lastLayerResult, val_labels, VAL_RATE * rows, lastHiddenLayerSize) << \
            ", Val Loss: " << crossEntropy(lastLayerResult, val_one_hot_labels, VAL_RATE * rows, lastHiddenLayerSize) << endl;
    }


    free(firstLayerResult);
    free(secondLayerResult);
    free(lastLayerResult);
    free(transposeFirstResult);
    free(transposeSecondResult);
    free(lastDelta);
    free(secondDelta);
    free(firstDelta);
    free(lastGradient);
    free(secondGradient);
    free(firstGradient);
    free(transposeLastWeight);
    free(transposeSecondWeight);
    free(reluDerivativeFirstMatrix);
    free(reluDerivativeSecondMatrix);
    free(transposeInputMatrix);
    free(thirdBiasGradient);
    free(secondBiasGradient);
    free(firstBiasGradient);
}

int main() {
    srand(static_cast<unsigned>(time(0)));
    // Data file
    string filename = "train-images-idx3-ubyte";
    string nameOfLabelFile = "train-labels-idx1-ubyte";

    // Input data
    double* input_data = NULL;
    double* input_labels = NULL;

    // Doc Du lieu
    unsigned int number_of_images, n_rows, n_cols;
    read_mnist(filename, input_data, number_of_images, n_rows, n_cols);

    unsigned int requiredMemsizeForLabel = number_of_images * 10;
    input_labels = (double*)malloc(requiredMemsizeForLabel * sizeof(double));

    for (int i = 0; i < requiredMemsizeForLabel; i++) {
      input_labels[i] = 0;
    }

    read_labels_one_hot(nameOfLabelFile, input_labels);

    // Cac ma tran trong so

    int inputLayerSize = 784;
    int firstHiddenLayerSize = 128;
    int secondHiddenLayerSize = 128;
    int lastHiddenLayerSize = 10;

    // Chia data
    double* train_data = NULL;
    double* train_one_hot_labels = NULL;
    double* train_labels = NULL;

    double* val_data = NULL;
    double* val_one_hot_labels = NULL;
    double* val_labels = NULL;

    double* test_data = NULL;
    double* test_one_hot_labels = NULL;
    double* test_labels = NULL;

    unsigned int sizeOfTrainData = TRAIN_RATE * number_of_images * inputLayerSize * sizeof(double);
    unsigned int sizeOfTrainOneHot = TRAIN_RATE * number_of_images * lastHiddenLayerSize * sizeof(double);
    unsigned int sizeOfTrainLabels = TRAIN_RATE * number_of_images * sizeof(double);

    unsigned int sizeOfValData = VAL_RATE * number_of_images * inputLayerSize * sizeof(double);
    unsigned int sizeOfValOneHot = VAL_RATE * number_of_images * lastHiddenLayerSize * sizeof(double);
    unsigned int sizeOfValLabels = VAL_RATE * number_of_images * sizeof(double);

    unsigned int sizeOfTestData = TEST_RATE * number_of_images * inputLayerSize * sizeof(double);
    unsigned int sizeOfTestOneHot = TEST_RATE * number_of_images * lastHiddenLayerSize * sizeof(double);
    unsigned int sizeOfTestLabels = TEST_RATE * number_of_images * sizeof(double);

    train_data = (double*)malloc(sizeOfTrainData);
    train_one_hot_labels = (double*)malloc(sizeOfTrainOneHot);
    train_labels = (double*)malloc(sizeOfTrainLabels);

    val_data = (double*)malloc(sizeOfValData);
    val_one_hot_labels = (double*)malloc(sizeOfValOneHot);
    val_labels = (double*)malloc(sizeOfValLabels);

    test_data = (double*)malloc(sizeOfTestData);
    test_one_hot_labels = (double*)malloc(sizeOfTestOneHot);
    test_labels = (double*)malloc(sizeOfTestLabels);

    // Khoi tao trong so
    unsigned int sizeOfFirstWeight = inputLayerSize * firstHiddenLayerSize;
    unsigned int sizeOfSecondWeight = firstHiddenLayerSize * secondHiddenLayerSize;
    unsigned int sizeOfLastWeight = secondHiddenLayerSize * lastHiddenLayerSize;

    double *firstHiddenLayerWeight = NULL;
    double *secondHiddenLayerWeight = NULL;
    double *lastHiddenLayerWeight = NULL;

    firstHiddenLayerWeight = (double*)malloc(sizeOfFirstWeight * sizeof(double));
    secondHiddenLayerWeight = (double*)malloc(sizeOfSecondWeight * sizeof(double));
    lastHiddenLayerWeight = (double*)malloc(sizeOfLastWeight * sizeof(double));

    initialize_weights(firstHiddenLayerWeight,inputLayerSize,firstHiddenLayerSize);
    initialize_weights(secondHiddenLayerWeight, firstHiddenLayerSize, secondHiddenLayerSize);
    initialize_weights(lastHiddenLayerWeight, secondHiddenLayerSize, lastHiddenLayerSize);

    // Khoi tao bias
    double* firstBiases = NULL;
    double* secondBiases = NULL;
    double* lastBiases = NULL;

    firstBiases = (double*)malloc(firstHiddenLayerSize * sizeof(double));
    secondBiases = (double*)malloc(secondHiddenLayerSize * sizeof(double));
    lastBiases = (double*)malloc(lastHiddenLayerSize * sizeof(double));

    initialize_biases(firstBiases, inputLayerSize, firstHiddenLayerSize);
    initialize_biases(secondBiases, firstHiddenLayerSize, secondHiddenLayerSize);
    initialize_biases(lastBiases, secondHiddenLayerSize, lastHiddenLayerSize);

    // Ma tran luu tru cac ket qua qua tung lop
    double* firstLayerResult = NULL;
    double* secondLayerResult = NULL;
    double* lastLayerResult = NULL;

    int sizeOfFirstLayerResult = number_of_images * firstHiddenLayerSize;
    int sizeOfSecondLayerResult = number_of_images * secondHiddenLayerSize;
    int sizeOfLastLayerResult = number_of_images * lastHiddenLayerSize;

    firstLayerResult = (double*)malloc(sizeOfFirstLayerResult * sizeof(double));
    secondLayerResult = (double*)malloc(sizeOfSecondLayerResult * sizeof(double));
    lastLayerResult = (double*)malloc(sizeOfLastLayerResult * sizeof(double));

    // Tao ma tran luu tru delta
    double* lastDelta = NULL;
    double* secondDelta = NULL;
    double* firstDelta = NULL;

    unsigned int lastDeltaSize = number_of_images * lastHiddenLayerSize;
    unsigned int secondDeltaSize = number_of_images * secondHiddenLayerSize;
    unsigned int firstDeltaSize = number_of_images * firstHiddenLayerSize;

    lastDelta = (double*)malloc(lastDeltaSize * sizeof(double));
    secondDelta = (double*)malloc(secondDeltaSize * sizeof(double));
    firstDelta = (double*)malloc(firstDeltaSize *sizeof(double));
    

    // Cap phat bo nho cho ma tran gradient
    double* lastGradient = NULL;
    double* secondGradient = NULL;
    double* firstGradient = NULL;

    lastGradient = (double*)malloc(sizeOfLastWeight * sizeof(double));
    secondGradient = (double*)malloc(sizeOfSecondWeight * sizeof(double));
    firstGradient = (double*)malloc(sizeOfFirstWeight * sizeof(double));

    // Cap phat bo nho cho ma tran chuyen vi
    
    double* transposedSecondResult = NULL;
    double* transposedFirstResult = NULL;
    double* transposedLastWeight = NULL;
    double* transposedSecondWeight = NULL;
    double* transposedInputMatrix = NULL;

    transposedSecondResult = (double*)malloc(sizeOfSecondLayerResult * sizeof(double));
    transposedFirstResult = (double*)malloc(sizeOfFirstLayerResult * sizeof(double));
    transposedLastWeight = (double*)malloc(sizeOfLastWeight * sizeof(double));
    transposedSecondWeight = (double*)malloc(sizeOfSecondWeight * sizeof(double));
    transposedInputMatrix = (double*)malloc((number_of_images * inputLayerSize) * sizeof(double));

    transposeMatrix(input_data, transposedInputMatrix, number_of_images, inputLayerSize);


    // Cap phat bo nho cho cac ma tran dao ham relu
    double* reluDerivativeSecondMatrix = NULL;
    double* reluDerivativeFirstMatrix = NULL;

    reluDerivativeSecondMatrix = (double*)malloc(sizeOfSecondLayerResult * sizeof(double));
    reluDerivativeFirstMatrix = (double*)malloc(sizeOfFirstLayerResult * sizeof(double));

    // Gradient cho bias
    double* firstBiasGradient = NULL;
    double* secondBiasGradient = NULL;
    double* thirdBiasGradient = NULL;

    firstBiasGradient = (double*)malloc(firstHiddenLayerSize * sizeof(double));
    secondBiasGradient = (double*)malloc(secondHiddenLayerSize * sizeof(double));
    thirdBiasGradient = (double*)malloc(lastHiddenLayerSize * sizeof(double));

    // Ground Truth Label
    double* labels = (double*)malloc(number_of_images*sizeof(double));
    read_labels(nameOfLabelFile, labels);

    shuffle_data(input_data, input_labels, labels, number_of_images, inputLayerSize, lastHiddenLayerSize);
    split(input_data, input_labels, labels, number_of_images, inputLayerSize, lastHiddenLayerSize, train_data, train_one_hot_labels, train_labels, val_data, val_one_hot_labels, val_labels, test_data, test_one_hot_labels, test_labels);

    cout << "Check label in train dataset\n\n";
    for (int label = 0; label < 10; label++) {
        unsigned int count = 0;
        for (int image = 0; image < TRAIN_RATE * number_of_images; image++) {
            if (train_labels[image] == label) {
                count++;
            }
        }

        cout << "Label: " << label <<", Count: " << count << endl;
    }

    cout << "\nCheck label in val dataset\n\n";
    for (int label = 0; label < 10; label++) {
        unsigned int count = 0;
        for (int image = 0; image < VAL_RATE * number_of_images; image++) {
            if (val_labels[image] == label) {
                count++;
            }
        }

        cout << "Label: " << label << ", count: " << count << endl;
    }

    cout << "\nCheck label in test dataset\n\n";
    for (int label = 0; label < 10; label++) {
        unsigned int count = 0;

        for (int image = 0; image < TEST_RATE * number_of_images; image++) {
            if (test_labels[image] == label) {
                count++;
            }
        }

        cout <<"Label: " << label << ", Count: " << count << endl;
    }
    int numTrainSamples = TRAIN_RATE * number_of_images;
    //===========================================================================================================================

    // for (int i = 0; i < NUM_EPOCH; i++) {
    //     // Forward qua 3 lop
    //     GpuTimer timer;
    //     timer.Start();

    //     forwardNN(train_data, firstHiddenLayerWeight, firstBiases, firstLayerResult, numTrainSamples, inputLayerSize, firstHiddenLayerSize);
    //     timer.Stop();
    //     float time = timer.Elapsed();
    //     printf("Thoi gian forward qua lop dau: %f \n", time);
    //     forwardNN(firstLayerResult, secondHiddenLayerWeight, secondBiases, secondLayerResult, numTrainSamples, firstHiddenLayerSize, secondHiddenLayerSize);
    //     forwardNN(secondLayerResult, lastHiddenLayerWeight, lastBiases, lastLayerResult, numTrainSamples, secondHiddenLayerSize, lastHiddenLayerSize, false);
    //     // Goi ham softmax cho ket qua cua layer cuoi
    //     timer.Start();
    //     softmax(lastLayerResult, numTrainSamples, lastHiddenLayerSize);
    //     timer.Stop();
    //     time = timer.Elapsed();
    //     printf("Thoi gian softmax qua lop dau: %f \n", time);

    //     // backprop

    //     // Tinh transpose truoc
    //     timer.Start();
    //     transposeMatrix(secondLayerResult, transposedSecondResult, numTrainSamples, secondHiddenLayerSize);
    //     timer.Stop();
    //     time = timer.Elapsed();
    //     printf("Thoi gian transpose thu hai: %f \n", time);
    //     timer.Start();
    //     transposeMatrix(firstLayerResult, transposedFirstResult, numTrainSamples, firstHiddenLayerSize);
    //     timer.Stop();
    //     time = timer.Elapsed();
    //     printf("Thoi gian transpose thu nhat: %f \n", time);

    //     timer.Start();
    //     calculateLastDelta(lastLayerResult, train_one_hot_labels, lastDelta, numTrainSamples, lastHiddenLayerSize);
    //     timer.Stop();
    //     time = timer.Elapsed();
    //     printf("Thoi gian tinh delta: %f \n", time);

    //     // Tinh cho gradient lop cuoi
    //     timer.Start();
    //     // multiplyMatrix(transposedSecondResult, lastDelta, lastGradient, secondHiddenLayerSize, numTrainSamples, lastHiddenLayerSize);
    //     // devideMatrixToScalar(lastGradient, numTrainSamples, secondHiddenLayerSize, lastHiddenLayerSize);
    //     backwardNN(transposedSecondResult, lastDelta, lastGradient, secondHiddenLayerSize, numTrainSamples, lastHiddenLayerSize);
    //     timer.Stop();
    //     time = timer.Elapsed();
    //     printf("Tinh gradient cho lop cuoi: %f \n", time);

    //     timer.Start();
    //     relu_derivative(secondLayerResult, reluDerivativeSecondMatrix, numTrainSamples, secondHiddenLayerSize);
    //     relu_derivative(firstLayerResult, reluDerivativeFirstMatrix, numTrainSamples, firstHiddenLayerSize);
    //     timer.Stop();
    //     time = timer.Elapsed();
    //     printf("Tong thoi gian tinh dao ham relu: %f \n", time);

    //     //gradientForBias(lastDelta, thirdBiasGradient, number_of_images, lastHiddenLayerSize);

    //     // Cho hidden layer 2

    //     //tinh delta
    //     timer.Start();
    //     transposeMatrix(lastHiddenLayerWeight, transposedLastWeight, secondHiddenLayerSize, lastHiddenLayerSize);
    //     multiplyMatrix(lastDelta, transposedLastWeight, secondDelta, numTrainSamples, lastHiddenLayerSize, secondHiddenLayerSize);
    //     multiplyMatrixElementWise(secondDelta, reluDerivativeSecondMatrix, secondDelta, numTrainSamples, secondHiddenLayerSize);
    //     timer.Stop();
    //     time = timer.Elapsed();
    //     printf("Thoi gian tinh delta 2: %f \n", time);

    //     //tinh gradient 
    //     timer.Start();
    //     // multiplyMatrix(transposedFirstResult, secondDelta, secondGradient, firstHiddenLayerSize, numTrainSamples, secondHiddenLayerSize);
    //     // devideMatrixToScalar(secondGradient, numTrainSamples, firstHiddenLayerSize, secondHiddenLayerSize);
    //     backwardNN(transposedFirstResult, secondDelta, secondGradient, firstHiddenLayerSize, numTrainSamples, secondHiddenLayerSize);
    //     timer.Stop();
    //     time = timer.Elapsed();
    //     printf("Thoi gian tinh gradient cho lop thu hai: %f \n", time);
    //     //gradientForBias(secondDelta, secondBiasGradient, number_of_images, secondHiddenLayerSize);

    //     // Cho hidden layer 1

    //     //tinh delta
    //     timer.Start();
    //     transposeMatrix(secondHiddenLayerWeight, transposedSecondWeight, firstHiddenLayerSize, secondHiddenLayerSize);
    //     multiplyMatrix(secondDelta, transposedSecondWeight, firstDelta, numTrainSamples, secondHiddenLayerSize, firstHiddenLayerSize);
    //     multiplyMatrixElementWise(firstDelta, reluDerivativeFirstMatrix, firstDelta, numTrainSamples, firstHiddenLayerSize);
    //     timer.Stop();
    //     time = timer.Elapsed();
    //     printf("Thoi gian tinh first delta: %f \n", time);

    //     // tinh gradient 
    //     // tinh chuyen vi cua ma tran dau vao
    //     timer.Start();
    //     // multiplyMatrix(transposedInputMatrix, firstDelta, firstGradient, inputLayerSize, numTrainSamples, firstHiddenLayerSize);
    //     // devideMatrixToScalar(firstGradient, numTrainSamples, inputLayerSize, firstHiddenLayerSize);
    //     backwardNN(transposedInputMatrix, firstDelta, firstGradient, inputLayerSize, numTrainSamples, firstHiddenLayerSize);
    //     timer.Stop();
    //     time = timer.Elapsed();
    //     printf("Thoi gian tinh gradient cho lop dau: %f \n", time);
    //     //gradientForBias(firstDelta, firstBiasGradient, number_of_images, firstHiddenLayerSize);

    //     timer.Start();
    //     // Cap nhat trong so
    //     // Layer 3
    //     updateWeights(lastHiddenLayerWeight, lastGradient, LEARNING_RATE, secondHiddenLayerSize, lastHiddenLayerSize);
    //     //updateBias(lastBiases, thirdBiasGradient, LEARNING_RATE,lastHiddenLayerSize);

    //     // Layer 2
    //     updateWeights(secondHiddenLayerWeight, secondGradient, LEARNING_RATE, firstHiddenLayerSize, secondHiddenLayerSize);
    //     //updateBias(secondBiases, secondBiasGradient, LEARNING_RATE,secondHiddenLayerSize);

    //     //layer 1
    //     updateWeights(firstHiddenLayerWeight, firstGradient, LEARNING_RATE, inputLayerSize, firstHiddenLayerSize);
    //     //updateBias(firstBiases, firstBiasGradient, LEARNING_RATE, firstHiddenLayerSize);
    //     timer.Stop();
    //     time = timer.Elapsed();
    //     cout << "Thoi gian cap nhat: " << time << " ms" << endl;

    //     forwardNN(train_data, firstHiddenLayerWeight, firstBiases, firstLayerResult, TRAIN_RATE * number_of_images, inputLayerSize, firstHiddenLayerSize);
    //     forwardNN(firstLayerResult, secondHiddenLayerWeight, secondBiases, secondLayerResult, TRAIN_RATE * number_of_images, firstHiddenLayerSize, secondHiddenLayerSize);
    //     forwardNN(secondLayerResult, lastHiddenLayerWeight, lastBiases, lastLayerResult, TRAIN_RATE * number_of_images, secondHiddenLayerSize, lastHiddenLayerSize, false);

    //     softmax(lastLayerResult, TRAIN_RATE * number_of_images, lastHiddenLayerSize);

    //     cout << "Epoch: " << i ;
    //     cout <<", Train Accuracy: " << accuracy(lastLayerResult, train_labels, TRAIN_RATE * number_of_images, lastHiddenLayerSize) << \
    //         ", Train Loss: " << crossEntropy(lastLayerResult, train_one_hot_labels, TRAIN_RATE * number_of_images, lastHiddenLayerSize);

    //     forwardNN(val_data, firstHiddenLayerWeight, firstBiases, firstLayerResult, VAL_RATE * number_of_images, inputLayerSize, firstHiddenLayerSize);
    //     forwardNN(firstLayerResult, secondHiddenLayerWeight, secondBiases, secondLayerResult, VAL_RATE * number_of_images, firstHiddenLayerSize, secondHiddenLayerSize);
    //     forwardNN(secondLayerResult, lastHiddenLayerWeight, lastBiases, lastLayerResult, VAL_RATE * number_of_images, secondHiddenLayerSize, lastHiddenLayerSize, false);

    //     softmax(lastLayerResult, VAL_RATE * number_of_images, lastHiddenLayerSize);

    //     cout <<", Val Accuracy: " << accuracy(lastLayerResult, val_labels, VAL_RATE * number_of_images, lastHiddenLayerSize) << \
    //         ", Val Loss: " << crossEntropy(lastLayerResult, val_one_hot_labels, VAL_RATE * number_of_images, lastHiddenLayerSize) << endl;
    // }
    trainNN(train_data, train_one_hot_labels, train_labels, val_data, val_labels, val_one_hot_labels, firstHiddenLayerWeight, secondHiddenLayerWeight, lastHiddenLayerWeight, firstBiases, secondBiases, lastBiases, NUM_EPOCH, number_of_images, inputLayerSize, firstHiddenLayerSize, secondHiddenLayerSize, lastHiddenLayerSize);
    //##############################################################################################################################################
    cout << endl;
    forwardNN(test_data, firstHiddenLayerWeight, firstBiases, firstLayerResult, TEST_RATE * number_of_images, inputLayerSize, firstHiddenLayerSize);
    forwardNN(firstLayerResult, secondHiddenLayerWeight, secondBiases, secondLayerResult, TEST_RATE * number_of_images, firstHiddenLayerSize, secondHiddenLayerSize);
    forwardNN(secondLayerResult, lastHiddenLayerWeight, lastBiases, lastLayerResult, TEST_RATE * number_of_images, secondHiddenLayerSize, lastHiddenLayerSize, false);

    // Goi ham softmax cho ket qua cua layer cuoi
    softmax(lastLayerResult, TEST_RATE * number_of_images, lastHiddenLayerSize);
    cout << "Accuracy: " << accuracy(lastLayerResult, test_labels, TEST_RATE * number_of_images, lastHiddenLayerSize);

    free(transposedInputMatrix);
    free(input_data);
    free(input_labels);
    free(firstHiddenLayerWeight);
    free(secondHiddenLayerWeight);
    free(lastHiddenLayerWeight);
    free(firstLayerResult);
    free(secondLayerResult);
    free(lastLayerResult);
    free(lastGradient);
    free(secondGradient);
    free(firstGradient);
    free(lastDelta);
    free(secondDelta);
    free(firstDelta);
    free(transposedSecondResult);
    free(transposedFirstResult);
    free(transposedLastWeight);
    free(transposedSecondWeight);
    free(reluDerivativeFirstMatrix);
    free(reluDerivativeSecondMatrix);
    free(firstBiases);
    free(secondBiases);
    free(lastBiases);
    free(firstBiasGradient);
    free(secondBiasGradient);
    free(thirdBiasGradient);
    free(labels);
    free(train_data);
    free(train_one_hot_labels);
    free(train_labels);
    free(val_data);
    free(val_one_hot_labels);
    free(val_labels);
    free(test_data);
    free(test_one_hot_labels);
    free(test_labels);
}